#include "hip/hip_runtime.h"
#include <sim_gpu.hpp>

#include <precision.hpp>
#include <math.h>

__device__
real dfdz2(const real *data, const int n, const int k, const int nZ, const int oodz2) {
  int in = n*nZ;

  return (data[k+1 + in] - 2.0f*data[k + in] + data[k-1 + in])*oodz2;
}

__device__
real dfdz(const real *data, const int n, const int k, const int nZ, const int oodz) {
  int in = n*nZ;

  return (data[k+1 + in] - data[k-1 + in])*oodz*0.5f;
}

__device__
real sqr(real x) {
  return x*x;
}

__global__
void gpu_computeLinearTemperatureDerivative(real *dTmpdt, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real oodz2) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN; n+=n_stride) {
    for(int k=1+k_index; k<nZ-1; k+=k_stride) {
      int i=k+n*nZ;
      dTmpdt[i] = dfdz2(tmp, n, k, nZ, oodz2) - sqr(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearVorticityDerivative(real *dOmgdt, const real *omg, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real Ra, const real Pr, const real oodz2) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN; n+=n_stride) {
    for(int k=1+k_index; k<nZ-1; k+=k_stride) {
      int i=k+n*nZ;
      dOmgdt[i] =
        Pr*(dfdz2(omg,n,k,nZ,oodz2) - sqr(n*M_PI/aspectRatio)*omg[i])
        + Pr*Ra*(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearXiDerivative(real *dXidt, const real *xi, real *dOmgdt, const real *omg,
    const int nN, const int nZ, const real tau, const real aspectRatio, const real RaXi, const real Pr, const real oodz2) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN; n+=n_stride) {
    for(int k=1+k_index; k<nZ-1; k+=k_stride) {
      int i=k+n*nZ;
      dXidt[i] = tau*(dfdz2(xi, n, k, nZ, oodz2) - pow(n*M_PI/aspectRatio, 2)*xi[i]);
      dOmgdt[i] += -RaXi*tau*Pr*(n*M_PI/aspectRatio)*xi[i];
    }
  }
}

__global__
void gpu_fillMode(real *data, const real value, const int n, const int nZ) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for(int k=index; k<nZ; k+=stride) {
    int i=k+n*nZ;
    data[i] = value;
  }
}

__global__
void gpu_addAdvectionApproximation(
    real *dVardt, const real *var,
    const real *psi,
    const int nN, const int nZ, const real aspectRatio, const real oodz) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=1+n_index; n<nN; n+=n_stride) {
    for(int k=1+k_index; k<nZ-1; k+=k_stride) {
      int i=k+n*nZ;
      dVardt[i] += -1*dfdz(var,0,k,nZ,oodz)*n*M_PI/aspectRatio * psi[i];
    }
  }
}

SimGPU::SimGPU(const Constants &c_in)
  : c(c_in)
  , vars(c_in)
  , keTracker(c_in)
{
  dt = c.initialDt;

  thomasAlgorithm = new ThomasAlgorithmGPU(c.nZ, c.nN, c.aspectRatio, c.oodz2);
}

SimGPU::~SimGPU() {
  delete thomasAlgorithm;
}

void SimGPU::computeLinearTemperatureDerivative() {
  dim3 threadsPerBlock(8,16);
  dim3 numBlocks((c.nN + 8 - 1)/8, (c.nZ - 2 + 16 - 1)/16);
  gpu_computeLinearTemperatureDerivative<<<numBlocks,threadsPerBlock>>>(vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), c.nN, c.nZ, c.aspectRatio, c.oodz2);
}

void SimGPU::computeLinearVorticityDerivative() {
  dim3 threadsPerBlock(8,16);
  dim3 numBlocks((c.nN + 8 - 1)/8, (c.nZ - 2 + 16 - 1)/16);
  gpu_computeLinearVorticityDerivative<<<numBlocks,threadsPerBlock>>>(vars.dOmgdt.getCurrent(), vars.omg.getCurrent(), vars.tmp.getCurrent(),
    c.nN, c.nZ, c.aspectRatio, c.Ra, c.Pr, c.oodz2);
}

void SimGPU::computeLinearXiDerivative() {
  dim3 threadsPerBlock(8,16);
  dim3 numBlocks((c.nN + 8 - 1)/8, (c.nZ - 2 + 16 - 1)/16);
  gpu_computeLinearXiDerivative<<<numBlocks,threadsPerBlock>>>(vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.dOmgdt.getCurrent(), vars.omg.getCurrent(),
    c.nN, c.nZ, c.tau, c.aspectRatio, c.RaXi, c.Pr, c.oodz2);
}

void SimGPU::computeLinearDerivatives() {
  // Computes the (linear) derivatives of Tmp and vars.omg
  computeLinearTemperatureDerivative();
  computeLinearVorticityDerivative();
  if(c.isDoubleDiffusion) {
    computeLinearXiDerivative();
  }
}

void SimGPU::addAdvectionApproximation() {
  // Only applies to the linear simulation
  dim3 threadsPerBlock(8,16);
  dim3 numBlocks((c.nN -1 + 8 - 1)/8, (c.nZ + 16 - 1)/16);
  gpu_fillMode<<<1,256>>>(vars.dOmgdt.getCurrent(), 0.0, 0, c.nZ);
  gpu_fillMode<<<1,256>>>(vars.dTmpdt.getCurrent(), 0.0, 0, c.nZ);
  gpu_addAdvectionApproximation<<<numBlocks,threadsPerBlock>>>(
      vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), vars.psi.getCurrent(),
      c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  if(c.isDoubleDiffusion) {
    gpu_fillMode<<<1,256>>>(vars.dXidt.getCurrent(), 0.0, 0, c.nZ);
    gpu_addAdvectionApproximation<<<1,1>>>(
        vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.psi.getCurrent(),
        c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  }
}

void SimGPU::solveForPsi(){
  // Solve for Psi using Thomas algorithm
  thomasAlgorithm->solve(vars.psi.getCurrent(), vars.omg.getCurrent());
}

void SimGPU::runLinearStep() {
  computeLinearDerivatives();
  addAdvectionApproximation();
  vars.updateVars(dt);
  vars.advanceDerivatives();
  solveForPsi();
}
