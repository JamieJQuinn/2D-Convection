#include "hip/hip_runtime.h"
#include <sim_gpu.hpp>

#include <precision.hpp>
#include <numerical_methods.hpp>
#include <complex_gpu.hpp>
#include <gpu_error_checking.hpp>

#include <math.h>
#include <iostream>

using std::cout;
using std::endl;

__device__ __constant__ int nX_d;
__device__ __constant__ int nN_d;
__device__ __constant__ int nZ_d;
__device__ __constant__ real oodz_d;
__device__ __constant__ real oodz2_d;
__device__ __constant__ real aspectRatio_d;
__device__ __constant__ real Ra_d;
__device__ __constant__ real Pr_d;
__device__ __constant__ real RaXi_d;
__device__ __constant__ real tau_d;

__device__
gpu_mode dfdz2(const gpu_mode *data, const int n, const int k) {
  return (data[calcIndex(n, k+1)] - 2.0f*data[calcIndex(n, k)] + data[calcIndex(n, k-1)])*oodz2_d;
}

__device__
gpu_mode dfdz(const gpu_mode *data, const int n, const int k) {
  return (data[calcIndex(n, k+1)] - data[calcIndex(n, k-1)])*oodz_d*0.5;
}

__device__
gpu_mode sqr(gpu_mode x) {
  return x*x;
}

__device__
gpu_mode sqr(real x) {
  return sqr(makeComplex(x, 0.0));
}

__global__
void gpu_computeLinearTemperatureDerivative(gpu_mode *dTmpdt, const gpu_mode *tmp) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN_d; n+=n_stride) {
    for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
      int i=calcIndex(n,k);
      dTmpdt[i] = dfdz2(tmp, n, k) - sqr(n*M_PI/aspectRatio_d)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearVorticityDerivative(gpu_mode *dOmgdt, const gpu_mode *omg, const gpu_mode *tmp) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN_d; n+=n_stride) {
    for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
      int i=calcIndex(n,k);
      dOmgdt[i] =
        Pr_d*(dfdz2(omg,n,k) - sqr(n*M_PI/aspectRatio_d)*omg[i])
        + Pr_d*Ra_d*(n*M_PI/aspectRatio_d)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearXiDerivative(gpu_mode *dXidt, const gpu_mode *xi, gpu_mode *dOmgdt, const gpu_mode *omg) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN_d; n+=n_stride) {
    for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
      int i=calcIndex(n,k);
      dXidt[i] = tau_d*(dfdz2(xi, n, k) - pow(n*M_PI/aspectRatio_d, 2)*xi[i]);
      dOmgdt[i] += -RaXi_d*tau_d*Pr_d*(n*M_PI/aspectRatio_d)*xi[i];
    }
  }
}

__global__
void gpu_fillMode(gpu_mode *data, const gpu_mode value, const int n) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for(int k=index; k<nZ_d; k+=stride) {
    int i=calcIndex(n,k);
    data[i] = value;
  }
}

__global__
void gpu_addAdvectionApproximation(
    gpu_mode *dVardt, const gpu_mode *var,
    const gpu_mode *psi) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=1+n_index; n<nN_d; n+=n_stride) {
    for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
      int i=calcIndex(n,k);
      dVardt[i] += -1*dfdz(var,0,k)*(n*M_PI/aspectRatio_d) * psi[i];
    }
  }
}

__global__
void gpu_computeNonlinearDerivativeN0(
    gpu_mode *dVardt, const gpu_mode *var,
    const gpu_mode *psi) {
  int k_index = blockIdx.x*blockDim.x + threadIdx.x;
  int k_stride = blockDim.x*gridDim.x;
  for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
    for(int n=1; n<nN_d; ++n) {
      // Contribution TO var[n=0]
      int i=calcIndex(n,k);
      dVardt[calcIndex(0,k)] +=
        -M_PI/(2*aspectRatio_d)*n*(
          dfdz(psi,n,k)*var[i] +
          dfdz(var,n,k)*psi[i]
          );
      // Contribution FROM var[n=0]
      dVardt[i] +=
        -n*M_PI/aspectRatio_d*psi[i]*dfdz(var,0,k);
    }
  }
}

__global__
void gpu_computeNonlinearDerivative(
    gpu_mode *dVardt, const gpu_mode *var,
    const gpu_mode *psi,
    const int vorticityFactor) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=1+n_index; n<nN_d; n+=n_stride) {
    // Contribution FROM var[n>0] and vars.omg[n>0]
    int o;
    for(int m=1; m<n; ++m){
      // Case n = n' + n''
      o = n-m;
      for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
        int im = calcIndex(m,k);
        int io = calcIndex(o,k);
        int in = calcIndex(n,k);
        dVardt[in] +=
          -M_PI/(2.0*aspectRatio_d)*(
          -m*dfdz(psi,o,k)*var[im]
          +o*dfdz(var,m,k)*psi[io]
          );
      }
    }
    for(int m=n+1; m<nN_d; ++m){
      // Case n = n' - n''
      o = m-n;
      for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
        int im = calcIndex(m,k);
        int io = calcIndex(o,k);
        int in = calcIndex(n,k);
        dVardt[in] +=
          -M_PI/(2.0*aspectRatio_d)*(
          +m*dfdz(psi,o,k)*var[im]
          +o*dfdz(var,m,k)*psi[io]
          );
      }
    }
    for(int m=1; m+n<nN_d; ++m){
      // Case n= n'' - n'
      o = n+m;
      for(int k=1+k_index; k<nZ_d-1; k+=k_stride) {
        int im = calcIndex(m,k);
        int io = calcIndex(o,k);
        int in = calcIndex(n,k);
        dVardt[in] +=
          vorticityFactor*M_PI/(2.0*aspectRatio_d)*(
          +m*dfdz(psi,o,k)*var[im]
          +o*dfdz(var,m,k)*psi[io]
          );
      }
    }
  }
}

SimGPU::SimGPU(const Constants &c_in)
  : c(c_in)
  , vars(c_in)
  , keTracker(c_in)
{
  dt = c.initialDt;

  thomasAlgorithm = new ThomasAlgorithmGPU(c.nZ, c.nN, c.aspectRatio, c.oodz2);

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nX_d), &c.nX, sizeof(c.nX), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nN_d), &c.nN, sizeof(c.nN), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nZ_d), &c.nZ, sizeof(c.nZ), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(oodz_d), &c.oodz, sizeof(c.oodz), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(oodz2_d), &c.oodz2, sizeof(c.oodz2), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(aspectRatio_d), &c.aspectRatio, sizeof(c.aspectRatio), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Ra_d), &c.Ra, sizeof(c.Ra), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(Pr_d), &c.Pr, sizeof(c.Pr), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), &c.tau, sizeof(c.tau), 0, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(RaXi_d), &c.RaXi, sizeof(c.RaXi), 0, hipMemcpyHostToDevice));
}

SimGPU::~SimGPU() {
  delete thomasAlgorithm;
}

void SimGPU::computeLinearTemperatureDerivative() {
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeLinearTemperatureDerivative<<<numBlocks,threadsPerBlock>>>(vars.dTmpdt.getCurrent(), vars.tmp.getCurrent());
}

void SimGPU::computeLinearVorticityDerivative() {
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeLinearVorticityDerivative<<<numBlocks,threadsPerBlock>>>(vars.dOmgdt.getCurrent(), vars.omg.getCurrent(), vars.tmp.getCurrent());
}

void SimGPU::computeLinearXiDerivative() {
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeLinearXiDerivative<<<numBlocks,threadsPerBlock>>>(vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.dOmgdt.getCurrent(), vars.omg.getCurrent());
}

void SimGPU::computeLinearDerivatives() {
  // Computes the (linear) derivatives of Tmp and vars.omg
  computeLinearTemperatureDerivative();
  computeLinearVorticityDerivative();
  if(c.isDoubleDiffusion) {
    computeLinearXiDerivative();
  }
}

void SimGPU::addAdvectionApproximation() {
  // Only applies to the linear simulation
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN - 1 + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  dim3 fillThreadsPerBlock(c.threadsPerBlock_x*c.threadsPerBlock_y);
  dim3 fillNumBlocks((c.nZ - 1 + fillThreadsPerBlock.x)/fillThreadsPerBlock.x);
  gpu_fillMode<<<fillNumBlocks,fillThreadsPerBlock>>>(vars.dOmgdt.getCurrent(), makeComplex(0.0, 0.0), 0);
  gpu_fillMode<<<fillNumBlocks,fillThreadsPerBlock>>>(vars.dTmpdt.getCurrent(), makeComplex(0.0, 0.0), 0);
  gpu_addAdvectionApproximation<<<numBlocks,threadsPerBlock>>>(
      vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), vars.psi.getCurrent());
  if(c.isDoubleDiffusion) {
    gpu_fillMode<<<fillNumBlocks,fillThreadsPerBlock>>>(vars.dXidt.getCurrent(), makeComplex(0.0, 0.0), 0);
    gpu_addAdvectionApproximation<<<numBlocks,threadsPerBlock>>>(
        vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.psi.getCurrent());
  }
}

void SimGPU::solveForPsi(){
  // Solve for Psi using Thomas algorithm
  thomasAlgorithm->solve(vars.psi.getCurrent(), vars.omg.getCurrent());
}

void SimGPU::runLinearStep() {
  computeLinearDerivatives();
  addAdvectionApproximation();
  vars.updateVars(dt);
  vars.advanceDerivatives();
  solveForPsi();
}

void SimGPU::computeNonlinearTemperatureDerivative() {
  // Calculate n=0 gpu_mode
  dim3 n0ThreadsPerBlock(c.threadsPerBlock_x*c.threadsPerBlock_y);
  dim3 n0NumBlocks((c.nZ - 1 + n0ThreadsPerBlock.x)/n0ThreadsPerBlock.x);
  gpu_computeNonlinearDerivativeN0<<<n0NumBlocks,n0ThreadsPerBlock>>>(vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), vars.psi.getCurrent());

  // Calculate other gpu_modes
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeNonlinearDerivative<<<numBlocks,threadsPerBlock>>>(vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), vars.psi.getCurrent(), -1);
}

void SimGPU::computeNonlinearXiDerivative() {
  // Calculate n=0 gpu_mode
  dim3 n0ThreadsPerBlock(c.threadsPerBlock_x*c.threadsPerBlock_y);
  dim3 n0NumBlocks((c.nZ - 1 + n0ThreadsPerBlock.x)/n0ThreadsPerBlock.x);
  gpu_computeNonlinearDerivativeN0<<<n0NumBlocks,n0ThreadsPerBlock>>>(vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.psi.getCurrent());

  // Calculate other gpu_modes
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeNonlinearDerivative<<<numBlocks,threadsPerBlock>>>(vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.psi.getCurrent(), -1);
}

void SimGPU::computeNonlinearVorticityDerivative() {
  dim3 threadsPerBlock(c.threadsPerBlock_x,c.threadsPerBlock_y);
  dim3 numBlocks((c.nN - 1 + threadsPerBlock.x - 1)/threadsPerBlock.x, (c.nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_computeNonlinearDerivative<<<numBlocks,threadsPerBlock>>>(vars.dOmgdt.getCurrent(), vars.omg.getCurrent(), vars.psi.getCurrent(), 1);
}

void SimGPU::runNonLinearStep(real f) {
  computeLinearDerivatives();
  computeNonlinearDerivatives();
  vars.updateVars(dt, f);
  vars.advanceDerivatives();
  solveForPsi();
}

void SimGPU::computeNonlinearDerivatives() {
  computeNonlinearTemperatureDerivative();
  computeNonlinearVorticityDerivative();
  if(c.isDoubleDiffusion) {
    computeNonlinearXiDerivative();
  }
}

void SimGPU::runNonLinear() {
  // Load initial conditions
  vars.load(c.icFile);

  real saveTime = 0;
  real KEcalcTime = 0;
  real KEsaveTime = 0;
  real CFLCheckTime = 0;
  real f = 1.0f; // Fractional change in dt (if CFL condition being breached)
  t = 0;
  while (c.totalTime-t>EPSILON) {
    //if(KEcalcTime-t < EPSILON) {
      //hipDeviceSynchronize();
      //keTracker.calcKineticEnergy(vars.psi);
      //KEcalcTime += 1e2*dt;
    //}
    //if(KEsaveTime-t < EPSILON) {
      //keTracker.saveKineticEnergy();
      //KEsaveTime += 1e4*dt;
    //}
    //if(CFLCheckTime-t < EPSILON) {
      //cout << "Checking CFL" << endl;
      //CFLCheckTime += 1e4*dt;
      //hipDeviceSynchronize();
      //f = checkCFL(vars.psi, c.dz, c.dx, dt, c.aspectRatio, c.nN, c.nX, c.nZ);
      //dt*=f;
    //}
    if(saveTime-t < EPSILON) {
      cout << t << " of " << c.totalTime << "(" << t/c.totalTime*100 << "%)" << endl;
      saveTime+=c.timeBetweenSaves;
      hipDeviceSynchronize();
      vars.save();
    }
    runNonLinearStep(f);
    t+=dt;
    f=1.0f;
  }
  hipDeviceSynchronize();
  printf("%e of %e (%.2f%%)\n", t, c.totalTime, t/c.totalTime*100);
  vars.save();
  //keTracker.saveKineticEnergy();
}
