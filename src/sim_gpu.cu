#include "hip/hip_runtime.h"
#include <sim_gpu.hpp>

#include <precision.hpp>
#include <math.h>

__device__
real dfdz2(const real *data, const int n, const int k, const int nZ, const int oodz2) {
  int in = n*nZ;

  return (data[k+1 + in] - 2.0f*data[k + in] + data[k-1 + in])*oodz2;
}

__device__
real dfdz(const real *data, const int n, const int k, const int nZ, const int oodz) {
  int in = n*nZ;

  return (data[k+1 + in] - data[k-1 + in])*oodz*0.5f;
}

__device__
real sqr(real x) {
  return x*x;
}

__global__
void gpu_computeLinearTemperatureDerivative(real *dTmpdt, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dTmpdt[i] = dfdz2(tmp, n, k, nZ, oodz2) - sqr(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearVorticityDerivative(real *dOmgdt, const real *omg, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real Ra, const real Pr, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dOmgdt[i] =
        Pr*(dfdz2(omg,n,k,nZ,oodz2) - sqr(n*M_PI/aspectRatio)*omg[i])
        + Pr*Ra*(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearXiDerivative(real *dXidt, const real *xi, real *dOmgdt, const real *omg,
    const int nN, const int nZ, const real tau, const real aspectRatio, const real RaXi, const real Pr, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dXidt[i] = tau*(dfdz2(xi, n, k, nZ, oodz2) - pow(n*M_PI/aspectRatio, 2)*xi[i]);
      dOmgdt[i] += -RaXi*tau*Pr*(n*M_PI/aspectRatio)*xi[i];
    }
  }
}

__global__
void gpu_fillMode(real *data, const real value, const int n, const int nZ) {
  for(int k=0; k<nZ; ++k) {
    int i=k+n*nZ;
    data[i] = value;
  }
}

__global__
void gpu_addAdvectionApproximation(
    real *dVardt, const real *var,
    const real *psi,
    const int nN, const int nZ, const real aspectRatio, const real oodz) {
  for(int n=1; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dVardt[i] += -1*dfdz(var,0,k,nZ,oodz)*n*M_PI/aspectRatio * psi[i];
    }
  }
}

SimGPU::SimGPU(const Constants &c_in)
  : c(c_in)
  , vars(c_in)
  , keTracker(c_in)
{
  dt = c.initialDt;

  thomasAlgorithm = new ThomasAlgorithmGPU(c.nZ, c.nN, c.aspectRatio, c.oodz2);
}

SimGPU::~SimGPU() {
  delete thomasAlgorithm;
}

void SimGPU::computeLinearTemperatureDerivative() {
  gpu_computeLinearTemperatureDerivative<<<1,1>>>(vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), c.nN, c.nZ, c.aspectRatio, c.oodz2);
}

void SimGPU::computeLinearVorticityDerivative() {
  gpu_computeLinearVorticityDerivative<<<1,1>>>(vars.dOmgdt.getCurrent(), vars.omg.getCurrent(), vars.tmp.getCurrent(),
    c.nN, c.nZ, c.aspectRatio, c.Ra, c.Pr, c.oodz2);
}

void SimGPU::computeLinearXiDerivative() {
  gpu_computeLinearXiDerivative<<<1,1>>>(vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.dOmgdt.getCurrent(), vars.omg.getCurrent(),
    c.nN, c.nZ, c.tau, c.aspectRatio, c.RaXi, c.Pr, c.oodz2);
}

void SimGPU::computeLinearDerivatives() {
  // Computes the (linear) derivatives of Tmp and vars.omg
  computeLinearTemperatureDerivative();
  computeLinearVorticityDerivative();
  if(c.isDoubleDiffusion) {
    computeLinearXiDerivative();
  }
}

void SimGPU::addAdvectionApproximation() {
  // Only applies to the linear simulation
  gpu_fillMode<<<1,1>>>(vars.dOmgdt.getCurrent(), 0.0, 0, c.nZ);
  gpu_fillMode<<<1,1>>>(vars.dTmpdt.getCurrent(), 0.0, 0, c.nZ);
  gpu_addAdvectionApproximation<<<1,1>>>(
      vars.dTmpdt.getCurrent(), vars.tmp.getCurrent(), vars.psi.getCurrent(),
      c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  if(c.isDoubleDiffusion) {
    gpu_fillMode<<<1,1>>>(vars.dXidt.getCurrent(), 0.0, 0, c.nZ);
    gpu_addAdvectionApproximation<<<1,1>>>(
        vars.dXidt.getCurrent(), vars.xi.getCurrent(), vars.psi.getCurrent(),
        c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  }
}

void SimGPU::solveForPsi(){
  // Solve for Psi using Thomas algorithm
  thomasAlgorithm->solve(vars.psi.getCurrent(), vars.omg.getCurrent());
}

void SimGPU::runLinearStep() {
  computeLinearDerivatives();
  addAdvectionApproximation();
  vars.updateVars(dt);
  vars.advanceDerivatives();
  solveForPsi();
}
