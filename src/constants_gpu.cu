#include <constants_gpu.hpp>
#include <gpu_error_checking.hpp>

__device__ __constant__ int nG_d;
__device__ __constant__ int nX_d;
__device__ __constant__ int nN_d;
__device__ __constant__ int nZ_d;
__device__ __constant__ real oodz_d;
__device__ __constant__ real oodx_d;
__device__ __constant__ real oodz2_d;
__device__ __constant__ real aspectRatio_d;
__device__ __constant__ real wavelength_d;
__device__ __constant__ gpu_mode xSinDerivativeFactor_d;
__device__ __constant__ gpu_mode xCosDerivativeFactor_d;
__device__ __constant__ real Ra_d;
__device__ __constant__ real Pr_d;
__device__ __constant__ real RaXi_d;
__device__ __constant__ real tau_d;

void copyConstantToGPU(const int &hostConstant, int &deviceConstant) {
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant), &hostConstant, sizeof(hostConstant), 0, hipMemcpyHostToDevice));
}

void copyConstantToGPU(const real &hostConstant, real &deviceConstant) {
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant), &hostConstant, sizeof(hostConstant), 0, hipMemcpyHostToDevice));
}

void copyConstantToGPU(const mode &hostConstant, gpu_mode &deviceConstant) {
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant), &hostConstant, sizeof(hostConstant), 0, hipMemcpyHostToDevice));
}


void copyGPUConstants(
    int nG, int nX, int nN, int nZ,
    real oodz, real oodx, real oodz2,
    real aspectRatio, real wavelength,
    mode xSinDerivativeFactor, mode xCosDerivativeFactor,
    real Ra, real Pr, real RaXi, real tau
  ) {
  copyConstantToGPU(nG, nG_d);
  copyConstantToGPU(nX, nX_d);
  copyConstantToGPU(nN, nN_d);
  copyConstantToGPU(nZ, nZ_d);
  copyConstantToGPU(oodz, oodz_d);
  copyConstantToGPU(oodx, oodx_d);
  copyConstantToGPU(oodz2, oodz2_d);
  copyConstantToGPU(aspectRatio, aspectRatio_d);
  copyConstantToGPU(wavelength, wavelength_d);
  copyConstantToGPU(xSinDerivativeFactor, xSinDerivativeFactor_d);
  copyConstantToGPU(xCosDerivativeFactor, xCosDerivativeFactor_d);
  copyConstantToGPU(Ra, Ra_d);
  copyConstantToGPU(Pr, Pr_d);
  copyConstantToGPU(tau, tau_d);
  copyConstantToGPU(RaXi, RaXi_d);
}
