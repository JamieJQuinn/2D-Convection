#include <complex_gpu.hpp>
#include <precision.hpp>

__device__ __host__ hipDoubleComplex  operator*(const hipDoubleComplex a, const hipDoubleComplex b) { return hipCmul(a,b); }
__device__ __host__ hipDoubleComplex  operator+(const hipDoubleComplex a, const hipDoubleComplex b) { return hipCadd(a,b); }
__device__ __host__ hipDoubleComplex  operator-(const hipDoubleComplex a, const hipDoubleComplex b) { return hipCsub(a,b); }
__device__ __host__ hipDoubleComplex  operator/(const hipDoubleComplex a, const hipDoubleComplex b) { return hipCdiv(a,b); }

__device__ __host__ hipComplex  operator*(const hipComplex a, const hipComplex b) { return hipCmulf(a,b); }
__device__ __host__ hipComplex  operator+(const hipComplex a, const hipComplex b) { return hipCaddf(a,b); }
__device__ __host__ hipComplex  operator-(const hipComplex a, const hipComplex b) { return hipCsubf(a,b); }
__device__ __host__ hipComplex  operator/(const hipComplex a, const hipComplex b) { return hipCdivf(a,b); }

__device__ __host__ gpu_mode& operator+=(gpu_mode& a, const gpu_mode b) {
  a = a+b;
  return a;
}

__device__ __host__ gpu_mode& operator-=(gpu_mode& a, const gpu_mode b) {
  a = a-b;
  return a;
}

__device__ __host__ gpu_mode makeComplex(real a, real b) {
#if defined PRECISION_DOUBLE
  return make_hipDoubleComplex(a, b);
#elif defined PRECISION_SINGLE
  return make_hipComplex(a, b);
#endif
}

__device__ __host__ gpu_mode  operator*(const real a, const gpu_mode b) {
#if defined PRECISION_DOUBLE
  return hipCmul(makeComplex(a, 0.0f), b);
#elif defined PRECISION_SINGLE
  return hipCmulf(makeComplex(a, 0.0f), b);
#endif
}

__device__ __host__ gpu_mode  operator*(const gpu_mode a, const real b) { return b*a; }


