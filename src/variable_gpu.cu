#include "hip/hip_runtime.h"
#include <variable_gpu.hpp>
#include <complex_gpu.hpp>
#include <precision.hpp>
#include <gpu_error_checking.hpp>

#include <iostream>

// CUDA constants
__device__ __constant__ int extern nG_d;
__device__ __constant__ int extern nX_d;
__device__ __constant__ int extern nN_d;
__device__ __constant__ int extern nZ_d;

__device__ int calcIndex(int n, int k) {
  return (k+nG_d)*(nX_d+2*nG_d) + n+nG_d;
}

__global__
void gpu_update(gpu_mode *var, const gpu_mode *dVardt, const gpu_mode *dVardtPrevious, const real dt, const real frac) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN_d; n+=n_stride) {
    for(int k=k_index; k<nZ_d; k+=k_stride) {
      int i=calcIndex(n, k);
      var[i] += ((2.0+frac)*dVardt[i] - frac*dVardtPrevious[i])*dt*0.5;
    }
  }
}

__global__
void normalise_fft(gpu_mode *data) {
  int n_index = blockIdx.x*blockDim.x + threadIdx.x;
  int n_stride = blockDim.x*gridDim.x;
  int k_index = blockIdx.y*blockDim.y + threadIdx.y;
  int k_stride = blockDim.y*gridDim.y;
  for(int n=n_index; n<nN_d; n+=n_stride) {
    for(int k=k_index; k<nZ_d; k+=k_stride) {
      int i=calcIndex(n, k);
      data[i] = data[i]*(1.0/nX_d);
    }
  }
}

__global__
void gpu_applyPeriodicHorizontalBoundaryConditions(real *data) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for(int k=index; k<nZ_d; k+=stride) {
    data[calcIndex(-1, k)] = data[calcIndex(nX_d-1,k)];
    data[calcIndex(nX_d, k)] = data[calcIndex(0,k)];
  }
}

void VariableGPU::applyVerticalBoundaryConditions() {
  if(verticalBoundaryConditions == BoundaryConditions::dirichlet) {
    int stripSize = (nG+1)*rowSize();
    gpuErrchk(hipMemcpy(data_d, bottomBoundary, stripSize*sizeof(bottomBoundary[0]), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(data_d+varSize()-stripSize, topBoundary, stripSize*sizeof(topBoundary[0]), hipMemcpyHostToDevice));
  } else if (verticalBoundaryConditions == BoundaryConditions::periodic) {
    // TODO copy top to bottom and bottom to top
  }
}

void VariableGPU::initialiseData(mode initialValue) {
  gpuErrchk(hipMalloc(&data_d, totalSize()*sizeof(gpu_mode)));
  gpuErrchk(hipMalloc(&spatialData_d, totalSize()*sizeof(real)));
  fill(initialValue);
}

void VariableGPU::fill(const mode value) {
  for(int i=0; i<this->totalSize(); ++i) {
    data[i] = value;
    spatialData[i] = value.real();
  }
  copyToDevice(true);
}

void VariableGPU::update(const VariableGPU& dVardt, const real dt, const real f) {
  dim3 threadsPerBlock(threadsPerBlock_x,threadsPerBlock_y);
  dim3 numBlocks((nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (nZ - 2 + threadsPerBlock.y - 1)/threadsPerBlock.y);
  gpu_update<<<numBlocks,threadsPerBlock>>>(this->getPlus(), dVardt.getCurrent(), dVardt.getPrevious(), dt, f);
}

void VariableGPU::readFromFile(std::ifstream& file) {
  Variable::readFromFile(file);
  copyToDevice();
  fillVerticalBoundaryConditions();
}

void VariableGPU::fillVerticalBoundaryConditions() {
  // Take all ghost rows and one regular row as boundary
  int stripSize = (nG+1)*rowSize();
  for(int i=0; i<stripSize; ++i) {
    bottomBoundary[i] = data[i];
    int dataIdx = varSize()-stripSize+i;
    topBoundary[i] = data[dataIdx];
  }
}

void VariableGPU::applyPhysicalHorizontalBoundaryConditions() {
  if(horizontalBoundaryConditions == BoundaryConditions::periodic) {
    dim3 threadsPerBlock(c.threadsPerBlock_x*c.threadsPerBlock_y);
    dim3 numBlocks((c.nZ - 1 + threadsPerBlock.x)/threadsPerBlock.x);
    gpu_applyPeriodicHorizontalBoundaryConditions<<<numBlocks,threadsPerBlock>>>(spatialData_d);
  }
}

//void VariableGPU::applyPhysicalVerticalBoundaryConditions() {
  //real nX = c.nX;
  //if(c.horizontalBoundaryConditions == BoundaryConditions::impermeable) {
    //for(int k=0; k<c.nZ; ++k) {
      //// Non-conducting
      //vars.tmp.spatial(-1,k) = vars.tmp.spatial(1, k);
      //vars.tmp.spatial(nX,k) = vars.tmp.spatial(nX-2, k);

      //// Impermeable
      //vars.psi.spatial(0,k) = 0.0;
      //vars.psi.spatial(nX-1,k) = 0.0;

      //// Stress free
      //vars.psi.spatial(-1,k) = 2.0*vars.psi.spatial(0,k) - vars.psi.spatial(1,k);
      //vars.psi.spatial(nX,k) = 2.0*vars.psi.spatial(nX-1,k) - vars.psi.spatial(nX-2,k);
      //vars.omg.spatial(0,k) = 0.0;
      //vars.omg.spatial(nX-1,k) = 0.0;
    //}
  //} else if(c.horizontalBoundaryConditions == BoundaryConditions::periodic) {
    //for(int k=0; k<c.nZ; ++k) {
      //vars.tmp.spatial(-1,k) = vars.tmp.spatial(nX-1, k);
      //vars.tmp.spatial(nX,k) = vars.tmp.spatial(0, k);

      //vars.omg.spatial(-1,k) = vars.omg.spatial(nX-1, k);
      //vars.omg.spatial(nX,k) = vars.omg.spatial(0, k);

      //vars.psi.spatial(-1,k) = vars.psi.spatial(nX-1, k);
      //vars.psi.spatial(nX,k) = vars.psi.spatial(0, k);
    //}
  //}
//}

void VariableGPU::writeToFile(std::ofstream& file) {
  copyToHost();
  Variable::writeToFile(file);
}

void VariableGPU::copyToDevice(bool copySpatial) {
  gpuErrchk(hipMemcpy(data_d, data, totalSize()*sizeof(data[0]), hipMemcpyHostToDevice));
  if(copySpatial) {
    gpuErrchk(hipMemcpy(spatialData_d, spatialData, totalSize()*sizeof(spatialData[0]), hipMemcpyHostToDevice));
  }
}

void VariableGPU::copyToHost(bool copySpatial) {
  gpuErrchk(hipMemcpy(data, data_d, totalSize()*sizeof(data[0]), hipMemcpyDeviceToHost));
  if(copySpatial) {
    gpuErrchk(hipMemcpy(spatialData, spatialData_d, totalSize()*sizeof(spatialData[0]), hipMemcpyDeviceToHost));
  }
}

VariableGPU::VariableGPU(const Constants &c_in, int totalSteps_in, bool useSinTransform_in):
  Variable(c_in, totalSteps_in)
  , data_d(nullptr)
  , spatialData_d(nullptr)
  , threadsPerBlock_x(c_in.threadsPerBlock_x)
  , threadsPerBlock_y(c_in.threadsPerBlock_y)
{
  initialiseData();
  setupcuFFT();
  topBoundary = new mode[(nG+1)*rowSize()];
  bottomBoundary = new mode[(nG+1)*rowSize()];
}

VariableGPU::~VariableGPU() {
  if(data_d != nullptr) {
    hipFree(data_d);
    data = nullptr;
  }
  if(spatialData_d != nullptr) {
    hipFree(spatialData_d);
    data = nullptr;
  }
  delete [] topBoundary;
  delete [] bottomBoundary;
}

gpu_mode* VariableGPU::getCurrent() {
  return (gpu_mode*)(getPlus(0));
}

const gpu_mode* VariableGPU::getCurrent() const {
  return (gpu_mode*)(getPlus(0));
}

gpu_mode* VariableGPU::getPrevious() {
  return (gpu_mode*)(data_d + previous*varSize());
}

const gpu_mode* VariableGPU::getPrevious() const {
  return (gpu_mode*)(data_d + previous*varSize());
}

gpu_mode* VariableGPU::getPlus(int nSteps) {
  return (gpu_mode*)(data_d + ((current+nSteps)%totalSteps)*varSize());
}

const gpu_mode* VariableGPU::getPlus(int nSteps) const {
  return (gpu_mode*)(data_d + ((current+nSteps)%totalSteps)*varSize());
}

void VariableGPU::setupcuFFT() {
  int rank = 1;
  int n[] = {nX};
  int inembed[] = {rowSize()};
  int istride = 1;
  int idist = rowSize();
  int onembed[] = {rowSize()};
  int ostride = 1;
  int odist = rowSize();
  int batch = nZ;

  hipfftType type = HIPFFT_D2Z;
  hipfftResult result = hipfftPlanMany(&cufftForwardPlan,
      rank, n,
      inembed, istride, idist,
      onembed, ostride, odist,
      type, batch);

  if(result != HIPFFT_SUCCESS) {
    std::cerr << "cuFFT forward plan could not be created" << std::endl;
  }

  type = HIPFFT_Z2D;
  result = hipfftPlanMany(&cufftBackwardPlan,
      rank, n,
      inembed, istride, idist,
      onembed, ostride, odist,
      type, batch);

  if(result != HIPFFT_SUCCESS) {
    std::cerr << "cuFFT backward plan could not be created" << std::endl;
  }
}

void VariableGPU::postFFTNormalise() {
  dim3 threadsPerBlock(threadsPerBlock_x,threadsPerBlock_y);
  dim3 numBlocks((nN + threadsPerBlock.x - 1)/threadsPerBlock.x, (nZ + threadsPerBlock.y - 1)/threadsPerBlock.y);
  normalise_fft<<<numBlocks,threadsPerBlock>>>(getCurrent());
}

void VariableGPU::toSpectral() {
  real *spatial = spatialData_d + calcIndex(0,0);
  gpu_mode *spectral = getCurrent() + calcIndex(0,0);

  hipfftResult result = hipfftExecD2Z(cufftForwardPlan, (hipfftDoubleReal*)spatial, (hipfftDoubleComplex*)spectral);

  if(result != HIPFFT_SUCCESS) {
    std::cerr << "cuFFT forward plan could not be executed. Error:" << result << std::endl;
  }

  postFFTNormalise();
  hipDeviceSynchronize();
}

void VariableGPU::toPhysical() {
  real *spatial = spatialData_d + calcIndex(0,0);
  gpu_mode *spectral = getCurrent() + calcIndex(0,0);

  hipfftResult result = hipfftExecZ2D(cufftBackwardPlan, (hipfftDoubleComplex*)spectral, (hipfftDoubleReal*)spatial);
  if(result != HIPFFT_SUCCESS) {
    std::cerr << "cuFFT backward plan could not be executed. Error:"<< result << std::endl;
  }
}
