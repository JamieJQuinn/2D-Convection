#include "hip/hip_runtime.h"
#include <variable_gpu.hpp>
#include <iostream>

__global__
void gpu_update(real *var, const real *dVardt, const real *dVardtPrevious, const real dt, const real frac,
    const int nN, const int nZ) {
  for(int n=0; n<nN; ++n) {
    for(int k=0; k<nZ; ++k) {
      int i=k+n*nZ;
      var[i] += ((2.0+frac)*dVardt[i] - frac*dVardtPrevious[i])*dt*0.5;
    }
  }
}

void VariableGPU::initialiseData(real initialValue) {
  hipMallocManaged(&data, totalSize()*sizeof(real));
  fill(initialValue);
}

void VariableGPU::update(const Variable& dVardt, const real dt, const real f) {
  gpu_update<<<1,1>>>(this->getPlus(), dVardt.getCurrent(), dVardt.getPrevious(), dt, f,
      nN, nZ);
}

VariableGPU::VariableGPU(const Constants &c_in, int totalSteps_in):
  Variable(c_in, totalSteps_in)
{}

VariableGPU::~VariableGPU() {
  if(data != NULL) {
    hipFree(data);
    data = NULL;
  }
}
