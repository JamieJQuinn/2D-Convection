#include "hip/hip_runtime.h"
#include <derivatives_gpu.hpp>

#include <precision.hpp>
#include <math.h>

__device__
real dfdz2(const real *data, const int n, const int k, const int nZ, const int oodz2) {
  int in = n*nZ;

  return (data[k+1 + in] - 2.0f*data[k + in] + data[k-1 + in])*oodz2;
}

__device__
real dfdz(const real *data, const int n, const int k, const int nZ, const int oodz) {
  int in = n*nZ;

  return (data[k+1 + in] - data[k-1 + in])*oodz*0.5f;
}

__device__
real sqr(real x) {
  return x*x;
}

__global__
void gpu_computeLinearTemperatureDerivative(real *dTmpdt, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dTmpdt[i] = dfdz2(tmp, n, k, nZ, oodz2) - sqr(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearVorticityDerivative(real *dOmgdt, const real *omg, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real Ra, const real Pr, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dOmgdt[i] =
        Pr*(dfdz2(omg,n,k,nZ,oodz2) - sqr(n*M_PI/aspectRatio)*omg[i])
        + Pr*Ra*(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_fillMode(real *data, const real value, const int n, const int nZ) {
  for(int k=0; k<nZ; ++k) {
    int i=k+n*nZ;
    data[i] = value;
  }
}

__global__
void gpu_addAdvectionApproximation(
    real *dVardt, const real *var,
    const real *psi,
    const int nN, const int nZ, const real aspectRatio, const real oodz) {
  for(int n=1; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dVardt[i] += -1*dfdz(var,0,k,nZ,oodz)*n*M_PI/aspectRatio * psi[i];
    }
  }
}

void computeLinearTemperatureDerivativeGPU(Variable &dTmpdt, const Variable &tmp, const Constants &c) {
  gpu_computeLinearTemperatureDerivative<<<1,1>>>(dTmpdt.data, tmp.data, c.nN, c.nZ, c.aspectRatio, c.oodz2);
}

void computeLinearVorticityDerivativeGPU(Variable &dOmgdt, const Variable &omg, const Variable &tmp, const Constants &c) {
  gpu_computeLinearVorticityDerivative<<<1,1>>>(dOmgdt.data, omg.data, tmp.data,
    c.nN, c.nZ, c.aspectRatio, c.Ra, c.Pr, c.oodz2);
}

void addAdvectionApproximationGPU(
    Variable &dTmpdt, const Variable &tmp,
    Variable &dOmgdt, const Variable &omg,
    Variable &dXidt, const Variable &xi,
    const Variable &psi,
    const Constants &c) {
  // Only applies to the linear simulation
  gpu_fillMode<<<1,1>>>(dOmgdt.data, 0.0, 0, c.nZ);
  gpu_fillMode<<<1,1>>>(dTmpdt.data, 0.0, 0, c.nZ);
  gpu_addAdvectionApproximation<<<1,1>>>(
      dTmpdt.data, tmp.data, psi.data,
      c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  if(c.isDoubleDiffusion) {
    gpu_fillMode<<<1,1>>>(dXidt.data, 0.0, 0, c.nZ);
    gpu_addAdvectionApproximation<<<1,1>>>(
        dXidt.data, xi.data, psi.data,
        c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  }
}
