#include "hip/hip_runtime.h"
#include <derivatives_gpu.hpp>

#include <precision.hpp>
#include <math.h>

__device__
real dfdz2(const real *data, const int n, const int k, const int nZ, const int oodz2) {
  int in = n*nZ;

  return (data[k+1 + in] - 2.0f*data[k + in] + data[k-1 + in])*oodz2;
}

__device__
real dfdz(const real *data, const int n, const int k, const int nZ, const int oodz) {
  int in = n*nZ;

  return (data[k+1 + in] - data[k-1 + in])*oodz*0.5f;
}

__device__
real sqr(real x) {
  return x*x;
}

__global__
void gpu_computeLinearTemperatureDerivative(real *dTmpdt, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dTmpdt[i] = dfdz2(tmp, n, k, nZ, oodz2) - sqr(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_computeLinearVorticityDerivative(real *dOmgdt, const real *omg, const real *tmp,
    const int nN, const int nZ, const real aspectRatio, const real Ra, const real Pr, const real oodz2) {
  for(int n=0; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dOmgdt[i] =
        Pr*(dfdz2(omg,n,k,nZ,oodz2) - sqr(n*M_PI/aspectRatio)*omg[i])
        + Pr*Ra*(n*M_PI/aspectRatio)*tmp[i];
    }
  }
}

__global__
void gpu_addAdvectionApproximation(
    real *dTmpdt, const real *tmp,
    real *dOmgdt, const real *omg,
    const real *psi,
    const int nN, const int nZ, const real aspectRatio, const real oodz) {
  // Only applies to the linear simulation
  for(int k=1; k<nZ-1; ++k) {
    int i=k+0*nZ;
    dOmgdt[i] = 0.0;
    dTmpdt[i] = 0.0;
  }
  for(int n=1; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dTmpdt[i] += -1*dfdz(tmp,0,k,nZ,oodz)*n*M_PI/aspectRatio * psi[i];
    }
  }
}

__global__ 
void gpu_addAdvectionApproximationToXi(
    real *dXidt, const real *xi,
    const real *psi,
    const int nN, const int nZ, const real aspectRatio, const real oodz) {
  for(int k=1; k<nZ-1; ++k) {
    int i=k+0*nZ;
    dXidt[i] = 0.0;
  }
  for(int n=1; n<nN; ++n) {
    for(int k=1; k<nZ-1; ++k) {
      int i=k+n*nZ;
      dXidt[i] += -1*dfdz(xi,0,k,nZ,oodz)*n*M_PI/aspectRatio * psi[i];
    }
  }
}

void computeLinearTemperatureDerivativeGPU(Variable &dTmpdt, const Variable &tmp, const Constants &c) {
  gpu_computeLinearTemperatureDerivative<<<1,1>>>(dTmpdt.data, tmp.data, c.nN, c.nZ, c.aspectRatio, c.oodz2);
}

void computeLinearVorticityDerivativeGPU(Variable &dOmgdt, const Variable &omg, const Variable &tmp, const Constants &c) {
  gpu_computeLinearVorticityDerivative<<<1,1>>>(dOmgdt.data, omg.data, tmp.data,
    c.nN, c.nZ, c.aspectRatio, c.Ra, c.Pr, c.oodz2);
}

void addAdvectionApproximationGPU(
    Variable &dTmpdt, const Variable &tmp,
    Variable &dOmgdt, const Variable &omg,
    Variable &dXidt, const Variable &xi,
    const Variable &psi,
    const Constants &c) {
  gpu_addAdvectionApproximation<<<1,1>>>(
      dTmpdt.data, tmp.data, dOmgdt.data, omg.data, psi.data,
      c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  if(c.isDoubleDiffusion) {
    gpu_addAdvectionApproximationToXi<<<1,1>>>(
        dXidt.data, xi.data, psi.data,
        c.nN, c.nZ, c.aspectRatio, 1.0f/c.dz);
  }
}
