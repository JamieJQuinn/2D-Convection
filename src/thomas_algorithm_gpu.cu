#include "hip/hip_runtime.h"
#include "thomas_algorithm_gpu.hpp"

__global__
void solveThomasAlgorithm(gpu_mode *sol, const gpu_mode *rhs, const real *wk1, const real *wk2, const real *sub, const int nN, const int nZ) {
  int mode = threadIdx.x;
  int stride = blockDim.x;
  for(int n=mode; n<nN; n+=stride) {
    int iN = n*nZ;

    // Forward Subsitution
    sol[0+iN] = rhs[0+iN]*wk1[0+iN];
    for (int i=1; i<nZ; ++i) {
      sol[i+iN] = (rhs[i+iN] - sub[i-1]*sol[i-1+iN])*wk1[i+iN];
    }
    // Backward Substitution
    for (int i=nZ-2; i>=0; --i) {
      sol[i+iN] -= wk2[i+iN]*sol[i+1+iN];
    }
  }
}

void ThomasAlgorithmGPU::solve(gpu_mode *sol, const gpu_mode *rhs) const {
  solveThomasAlgorithm<<<1,256>>>((gpu_mode*)sol, (gpu_mode*)rhs, (real*)wk1, (real*)wk2, (real*)sub, nN, nZ);
}

void ThomasAlgorithmGPU::formTriDiagonalArraysForN (
          const real *sub, const real *dia, const real *sup,
    real * wk1, real *wk2) {

  wk1[0] = 1.0/dia[0];
  wk2[0] = sup[0]*wk1[0];

  for (int i=1; i<nZ-1; ++i) {
    wk1[i] = 1.0/(dia[i] - sub[i-1] * wk2[i-1]);
    wk2[i] = sup[i]*wk1[i];
  }

  wk1[nZ-1] = 1.0/(dia[nZ-1] - sub[nZ-2]*wk2[nZ-2]);
}

ThomasAlgorithmGPU::ThomasAlgorithmGPU(const int nZ, const int nN, const int a, const real oodz2):
  nZ(nZ),
  nN(nN),
  oodz2(oodz2)
  {
  hipMallocManaged(&wk1, nZ*nN*sizeof(real));
  hipMallocManaged(&wk2, nZ*nN*sizeof(real));
  hipMallocManaged(&sub, nZ*sizeof(real));

  // Precalculate tridiagonal arrays
  real * dia = new real [nZ];
  real * sup = new real [nZ];
  for(int k=0; k<nZ; ++k) {
    sub[k] = sup[k] = -oodz2;
  }
  for(int n=0; n<nN; ++n) {
    for(int k=0; k<nZ; ++k){
      dia[k] = pow(M_PI/a*n, 2) + 2*oodz2;
    }
    dia[0] = dia[nZ-1] = 1.0;
    sub[nZ-2] = sup[0] = 0.0;
    formTriDiagonalArraysForN(
    sub, dia, sup,
    wk1+n*nZ, wk2+n*nZ);
  }

  delete [] dia;
  delete [] sup;
}

ThomasAlgorithmGPU::~ThomasAlgorithmGPU() {
  hipFree(wk1);
  hipFree(wk2);
  hipFree(sub);
}
