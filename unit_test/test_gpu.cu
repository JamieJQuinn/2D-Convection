#include "catch.hpp"
#include "thomas_algorithm_gpu.hpp"
#include "thomas_algorithm.hpp"
#include "constants.hpp"
#include "variable.hpp"
#include "variable_gpu.hpp"
#include "sim.hpp"
#include "sim_gpu.hpp"

#include <iostream>

using namespace std;

TEST_CASE( "GPU loads and prints values correctly", "[gpu]" ) {
  int nZ = 5;

  real *rhsGPU;
  real *solGPU;

  hipMallocManaged(&rhsGPU, nZ*sizeof(real));
  hipMallocManaged(&solGPU, nZ*sizeof(real));

  for(int i=0; i<nZ; ++i) {
    rhsGPU[i] = i+1;
    solGPU[i] = 0.0;
  }

  hipDeviceSynchronize();

  for(int i=0; i<nZ; ++i) {
    REQUIRE(rhsGPU[i] == real(i+1));
  }
}

TEST_CASE( "GPU Thomas algorithm solves a system correctly", "[gpu]" ) {
  int nZ = 10;
  int nN = 5;

  ThomasAlgorithm ta (nZ, nN, 1, 0.01f);
  ThomasAlgorithmGPU taGPU (nZ, nN, 1, 0.01f);

  real rhs [nZ];
  real sol [nZ];

  real *rhsGPU;
  real *solGPU;

  hipMallocManaged(&rhsGPU, nN*nZ*sizeof(real));
  hipMallocManaged(&solGPU, nN*nZ*sizeof(real));

  for(int i=0; i<nZ; ++i) {
    rhsGPU[i+2*nZ] = i+1;
    solGPU[i+2*nZ] = 0.0;
    rhs[i] = i+1;
    sol[i] = 0.0;
  }

  // Check precalculation works

  for(int i=0; i<nZ; ++i) {
    REQUIRE(ta.sub[i] == Approx(taGPU.sub[i]));
  }

  for(int n=0; n<nN; ++n) {
    for(int k=0; k<nZ; ++k) {
      REQUIRE(ta.wk1[k+n*nZ] == Approx(taGPU.wk1[k+n*nZ]));
      if(k < nZ-1){
        REQUIRE(ta.wk2[k+n*nZ] == Approx(taGPU.wk2[k+n*nZ]));
      }
    }
  }

  ta.solve((real*)sol, (real*)rhs, 2);
  taGPU.solve((real*)solGPU, (real*)rhsGPU);

  hipDeviceSynchronize();

  for(int i=0; i<nZ; ++i) {
    CHECK(solGPU[i+2*nZ] == Approx(sol[i]));
    CHECK(solGPU[i+0*nZ] == Approx(0.0));
    CHECK(solGPU[i+1*nZ] == Approx(0.0));
    CHECK(solGPU[i+3*nZ] == Approx(0.0));
    CHECK(solGPU[i+4*nZ] == Approx(0.0));
  }
}

TEST_CASE("GPU variable works", "[gpu]") {
  Constants c;
  c.nN = 5;
  c.nZ = 10;
  c.aspectRatio = 1;
  c.calculateDerivedConstants();

  // Create GPU variables
  VariableGPU tmp(c);
  tmp.initialiseData();

  tmp.data[5] = 1.0f;

  REQUIRE(tmp.data[5] == Approx(1.0f));
}

TEST_CASE("SimGPU initialises OK", "[gpu]") {
  Constants c;
  c.nN = 5;
  c.nZ = 10;
  c.aspectRatio = 1;
  c.calculateDerivedConstants();

  // Create GPU variables
  SimGPU sim(c);

  sim.vars.reinit(2.0);

  sim.vars.tmp(0,0) = 1.0;

  REQUIRE(sim.vars.tmp(0,0) == Approx(1.0));
  REQUIRE(sim.vars.tmp(0,1) == Approx(2.0));
}

TEST_CASE("Linear step calculates correctly", "[gpu]") {
  Constants c;
  c.nN = 5;
  c.nZ = 10;
  c.aspectRatio = 1.3;
  c.Pr = 1.0;
  c.Ra = 2.5;
  c.RaXi = 2.0;
  c.tau = 0.01;
  c.isDoubleDiffusion = true;
  c.calculateDerivedConstants();

  Sim s(c);
  SimGPU sGPU(c);

  // Load both with same test data
  for(int n=0; n<c.nN; ++n) {
    for(int k=0; k<c.nZ; ++k) {
      s.vars.omg(n,k) = (float)k;
      s.vars.tmp(n,k) = (float)k/c.nZ;
      s.vars.psi(n,k) = (float)k/c.nN;
      s.vars.xi(n,k) = (float)k/c.nN;
    }
  }

  for(int n=0; n<c.nN; ++n) {
    for(int k=0; k<c.nZ; ++k) {
      sGPU.vars.omg(n,k) = (float)k;
      sGPU.vars.tmp(n,k) = (float)k/c.nZ;
      sGPU.vars.psi(n,k) = (float)k/c.nN;
      sGPU.vars.xi(n,k) = (float)k/c.nN;
    }
  }

  s.runLinearStep();
  sGPU.runLinearStep();

  hipDeviceSynchronize();

  for(int n=0; n<c.nN; ++n) {
    for(int k=1; k<c.nZ-1; ++k) {
      CHECK(sGPU.vars.dOmgdt(n,k) == Approx(s.vars.dOmgdt(n,k)));
      CHECK(sGPU.vars.dTmpdt(n,k) == Approx(s.vars.dTmpdt(n,k)));
      CHECK(sGPU.vars.dXidt(n,k) == Approx(s.vars.dXidt(n,k)));
      CHECK(sGPU.vars.tmp(n,k) == Approx(s.vars.tmp(n,k)));
      CHECK(sGPU.vars.psi(n,k) == Approx(s.vars.psi(n,k)));
      CHECK(sGPU.vars.omg(n,k) == Approx(s.vars.omg(n,k)));
      CHECK(sGPU.vars.xi(n,k) == Approx(s.vars.xi(n,k)));
    }
  }
}
